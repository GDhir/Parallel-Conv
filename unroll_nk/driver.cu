
#include <hip/hip_runtime.h>
// #include <iostream>
// #include <cuda.h>
// #include <stdio.h>
//you can change the grid_size
//you can change the block_size
#define BLOCK_SIZE 128
__global__ void cnn(int N,int C,int K,int H,int W,int R, int S, int u, int v, int P, int Q,
               float *d_input, float * d_weight, float * d_output){
    //@@ cnn kernel design

    int idx = threadIdx.x + blockIdx.x*BLOCK_SIZE;

    // long int Nout = N*K*P*Q;

    if( idx < N*K*P*Q/16 ) {
        int q = idx%Q;
        int p = (idx/Q)%P;
        int k = ( ( 4*(idx/Q/P) )%K );
        int n = 4*( ( 4*(idx/Q/P) )/K );

        int ij = p*u;
        int ii = q*v;

        float sumval1 = 0;
        float sumval2 = 0;
        float sumval3 = 0;
        float sumval4 = 0;
        float sumval5 = 0;
        float sumval6 = 0;
        float sumval7 = 0;
        float sumval8 = 0;
        float sumval9 = 0;
        float sumval10 = 0;
        float sumval11 = 0;
        float sumval12 = 0;
        float sumval13 = 0;
        float sumval14 = 0;
        float sumval15 = 0;
        float sumval16 = 0;

            for (unsigned int r = 0; r<R; r++) { // filter height
                for (unsigned int s = 0; s < S; s++) {// filter width
                    for(unsigned int c=0; c<C; c++) { 
                    //output_seq[n][k][p][q] += input [n][c][ij+r][ii+s] * weight[k][c][r][s];
                    sumval1 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                    sumval2 += d_input[(n)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+c*R*S+r*S+s];
                    sumval3 += d_input[(n)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 2)*C*R*S+c*R*S+r*S+s];
                    sumval4 += d_input[(n)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 3)*C*R*S+c*R*S+r*S+s];
                    sumval5 += d_input[(n + 1)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k)*C*R*S+c*R*S+r*S+s];
                    sumval6 += d_input[(n + 1)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+c*R*S+r*S+s];
                    sumval7 += d_input[(n + 1)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 2)*C*R*S+c*R*S+r*S+s];
                    sumval8 += d_input[(n + 1)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 3)*C*R*S+c*R*S+r*S+s];
                    sumval9 += d_input[(n + 2)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k)*C*R*S+c*R*S+r*S+s];
                    sumval10 += d_input[(n + 2)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+c*R*S+r*S+s];
                    sumval11 += d_input[(n + 2)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 2)*C*R*S+c*R*S+r*S+s];
                    sumval12 += d_input[(n + 2)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 3)*C*R*S+c*R*S+r*S+s];
                    sumval13 += d_input[(n + 3)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k)*C*R*S+c*R*S+r*S+s];
                    sumval14 += d_input[(n + 3)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+c*R*S+r*S+s];
                    sumval15 += d_input[(n + 3)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 2)*C*R*S+c*R*S+r*S+s];
                    sumval16 += d_input[(n + 3)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 3)*C*R*S+c*R*S+r*S+s];

                }
            }
        }

        d_output[n*K*P*Q + k*P*Q + p*Q + q] = sumval1;
        d_output[(n)*K*P*Q + (k + 1)*P*Q + p*Q + q] = sumval2;
        d_output[(n)*K*P*Q + (k + 2)*P*Q + p*Q + q] = sumval3;
        d_output[(n)*K*P*Q + (k + 3)*P*Q + p*Q + q] = sumval4;
        d_output[(n + 1)*K*P*Q + (k)*P*Q + p*Q + q] = sumval5;
        d_output[(n + 1)*K*P*Q + (k + 1)*P*Q + p*Q + q] = sumval6;
        d_output[(n + 1)*K*P*Q + (k + 2)*P*Q + p*Q + q] = sumval7;
        d_output[(n + 1)*K*P*Q + (k + 3)*P*Q + p*Q + q] = sumval8;
        d_output[(n + 2)*K*P*Q + (k)*P*Q + p*Q + q] = sumval9;
        d_output[(n + 2)*K*P*Q + (k + 1)*P*Q + p*Q + q] = sumval10;
        d_output[(n + 2)*K*P*Q + (k + 2)*P*Q + p*Q + q] = sumval11;
        d_output[(n + 2)*K*P*Q + (k + 3)*P*Q + p*Q + q] = sumval12;
        d_output[(n + 3)*K*P*Q + (k)*P*Q + p*Q + q] = sumval13;
        d_output[(n + 3)*K*P*Q + (k + 1)*P*Q + p*Q + q] = sumval14;
        d_output[(n + 3)*K*P*Q + (k + 2)*P*Q + p*Q + q] = sumval15;
        d_output[(n + 3)*K*P*Q + (k + 3)*P*Q + p*Q + q] = sumval16;

    }

}
