
#include <hip/hip_runtime.h>
// #include <iostream>
// #include <cuda.h>
// #include <stdio.h>
//you can change the grid_size
//you can change the block_size
#define BLOCK_SIZE 128
__global__ void cnn(int N,int C,int K,int H,int W,int R, int S, int u, int v, int P, int Q,
               float *d_input, float * d_weight, float * d_output){
    //@@ cnn kernel design

    int idx = threadIdx.x + blockIdx.x*BLOCK_SIZE;

    // long int Nout = N*K*P*Q;

    if( idx < N*K*P*Q ) {
        int q = idx%Q;
        int p = (idx/Q)%P;
        int k = (idx/Q/P)%K;
        int n = idx/Q/P/K;

        int ij = p*u;
        int ii = q*v;

        float sumval = 0;
        int remc{ C%16 };

        // for(unsigned int c=0; c<C; c++) { // input feature map
        for (unsigned int r = 0; r<R; r++) { // filter height
            for (unsigned int s = 0; s < S; s++) {// filter width

                for( unsigned int c = 0; c < remc; c++ ) {
                    sumval += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                }

                for(unsigned int c = remc; c < C; c += 16) {
            //output_seq[n][k][p][q] += input [n][c][ij+r][ii+s] * weight[k][c][r][s];
                    sumval += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                    sumval += d_input[n*C*H*W + (c + 1)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 1)*R*S + r*S + s];
                    sumval += d_input[n*C*H*W + (c + 2)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 2)*R*S+r*S+s];
                    sumval += d_input[n*C*H*W + (c + 3)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 3)*R*S + r*S + s];
                    sumval += d_input[n*C*H*W + (c + 4)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 4)*R*S+r*S+s];
                    sumval += d_input[n*C*H*W + (c + 5)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 5)*R*S + r*S + s];
                    sumval += d_input[n*C*H*W + (c + 6)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 6)*R*S+r*S+s];
                    sumval += d_input[n*C*H*W + (c + 7)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 7)*R*S + r*S + s];
                    sumval += d_input[n*C*H*W + (c + 8)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 8)*R*S + r*S + s];
                    sumval += d_input[n*C*H*W + (c + 9)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 9)*R*S+r*S+s];
                    sumval += d_input[n*C*H*W + (c + 10)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 10)*R*S + r*S + s];
                    sumval += d_input[n*C*H*W + (c + 11)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 11)*R*S+r*S+s];
                    sumval += d_input[n*C*H*W + (c + 12)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 12)*R*S + r*S + s];
                    sumval += d_input[n*C*H*W + (c + 13)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 13)*R*S+r*S+s];
                    sumval += d_input[n*C*H*W + (c + 14)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 14)*R*S + r*S + s];
                    sumval += d_input[n*C*H*W + (c + 15)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 15)*R*S + r*S + s];
                }
            }
        }

        // for (unsigned int r = 0; r<R; r++) { // filter height
        //     for (unsigned int s = 0; s < S; s++) {// filter width

        //         for( unsigned int cb = 0; cb < C; cb += BLOCK_SIZE ) {

        //             for(unsigned int c = cb; c < min( cb + BLOCK_SIZE, C ); c++) {
        //             //output_seq[n][k][p][q] += input [n][c][ij+r][ii+s] * weight[k][c][r][s];
        //                 sumval += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
        //             }

        //         }
        //     }
        // }

        d_output[idx] = sumval;

        // # if __CUDA_ARCH__>=200
        //     // if( n == 63 )
        //     //     printf("%f \t %d \t %d \t %d \t %d \n", sumval, q, p, k, n);

        //     if( idx == 0 )
        //         printf( "%d \n", blockDim.x*gridDim.x );
        // #endif  
    }

}
