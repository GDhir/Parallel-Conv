
#include <hip/hip_runtime.h>
// #include <iostream>
// #include <cuda.h>
// #include <stdio.h>
//you can change the grid_size
//you can change the block_size
#define BLOCK_SIZE 128
__global__ void cnn(int N,int C,int K,int H,int W,int R, int S, int u, int v, int P, int Q,
               float *d_input, float * d_weight, float * d_output){
    //@@ cnn kernel design

    int idx = threadIdx.x + blockIdx.x*BLOCK_SIZE;

    // long int Nout = N*K*P*Q;

    if( idx < N*K*P*Q ) {
        int q = idx%Q;
        int p = (idx/Q)%P;
        int k = (idx/Q/P)%K;
        int n = idx/Q/P/K;

        int ij = p*u;
        int ii = q*v;

        float sumval = 0;

        // for(unsigned int c=0; c<C; c++) { // input feature map
        for (unsigned int r = 0; r<R; r++) { // filter height
            for (unsigned int s = 0; s < S; s++) {// filter width
                for(unsigned int c=0; c<C; c++) {

            //output_seq[n][k][p][q] += input [n][c][ij+r][ii+s] * weight[k][c][r][s];
                    sumval += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                }
            }
        }

        d_output[idx] = sumval;
    }

}
