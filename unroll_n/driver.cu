
#include <hip/hip_runtime.h>
// #include <iostream>
// #include <cuda.h>
// #include <stdio.h>
//you can change the grid_size
//you can change the block_size
#define BLOCK_SIZE 1024
__global__ void cnn(int N,int C,int K,int H,int W,int R, int S, int u, int v, int P, int Q,
               float *d_input, float * d_weight, float * d_output){
    //@@ cnn kernel design

    int idx = threadIdx.x + blockIdx.x*BLOCK_SIZE;

    // long int Nout = N*K*P*Q;

    if( idx < N*K*P*Q/4 ) {
        int q = idx%Q;
        int p = (idx/Q)%P;
        int k = (idx/Q/P)%K;
        int n = 4*(idx/Q/P/K);

        int ij = p*u;
        int ii = q*v;

        float sumval1 = 0;
        float sumval2 = 0;
        float sumval3 = 0;
        float sumval4 = 0;

        for(unsigned int c=0; c<C; c++) { // input feature map
            for (unsigned int r = 0; r<R; r++) { // filter height
                for (unsigned int s = 0; s < S; s++) {// filter width
                    //output_seq[n][k][p][q] += input [n][c][ij+r][ii+s] * weight[k][c][r][s];
                    sumval1 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                    sumval2 += d_input[(n + 1)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                    sumval3 += d_input[(n + 2)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                    sumval4 += d_input[(n + 3)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                }
            }
        }
        d_output[n*K*P*Q + k*P*Q + p*Q + q] = sumval1;
        d_output[(n + 1)*K*P*Q + k*P*Q + p*Q + q] = sumval2;
        d_output[(n + 2)*K*P*Q + k*P*Q + p*Q + q] = sumval3;
        d_output[(n + 3)*K*P*Q + k*P*Q + p*Q + q] = sumval4;

        // # if __CUDA_ARCH__>=200
        //     // if( n == 63 )
        //     //     printf("%f \t %d \t %d \t %d \t %d \n", sumval, q, p, k, n);

        //     if( idx == 0 )
        //         printf( "%d \n", blockDim.x*gridDim.x );
        // #endif  
    }

}
