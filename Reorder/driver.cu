
#include <hip/hip_runtime.h>
// #include <iostream>
// #include <cuda.h>
// #include <stdio.h>
//you can change the grid_size
//you can change the block_size
#define BLOCK_SIZE 256
__global__ void cnn(int N,int C,int K,int H,int W,int R, int S, int u, int v, int P, int Q,
               float *d_input, float * d_weight, float * d_output){
    //@@ cnn kernel design

    int idx = threadIdx.x + blockIdx.x*BLOCK_SIZE;

    // long int Nout = N*K*P*Q;

    if( idx < N*K*P*Q ) {
        int q = idx%Q;
        int p = (idx/Q)%P;
        int n = (idx/Q/P)%N;
        int k = idx/Q/P/N;

        int ij = p*u;
        int ii = q*v;

        float sumval = 0;

        for(unsigned int c=0; c<C; c++) { // input feature map
            for (unsigned int r = 0; r<R; r++) { // filter height
                for (unsigned int s = 0; s < S; s++) {// filter width
                    //output_seq[n][k][p][q] += input [n][c][ij+r][ii+s] * weight[k][c][r][s];
                    sumval += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                }
            }
        }
        d_output[n*K*P*Q + k*P*Q + p*Q + q] = sumval;

        // # if __CUDA_ARCH__>=200
        //     // if( n == 63 )
        //     //     printf("%f \t %d \t %d \t %d \t %d \n", sumval, q, p, k, n);

        //     if( idx == 0 )
        //         printf( "%d \n", blockDim.x*gridDim.x );
        // #endif  
    }

}
