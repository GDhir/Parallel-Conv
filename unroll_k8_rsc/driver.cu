
#include <hip/hip_runtime.h>
// #include <iostream>
// #include <cuda.h>
// #include <stdio.h>
//you can change the grid_size
//you can change the block_size
#define BLOCK_SIZE 128
__global__ void cnn(int N,int C,int K,int H,int W,int R, int S, int u, int v, int P, int Q,
               float *d_input, float * d_weight, float * d_output){
    //@@ cnn kernel design

    int idx = threadIdx.x + blockIdx.x*BLOCK_SIZE;

    // long int Nout = N*K*P*Q;

    if( idx < N*K*P*Q/8 ) {
        int q = idx%Q;
        int p = (idx/Q)%P;
        int k = ( ( 8*(idx/Q/P) )%K );
        int n = ( 8*(idx/Q/P) )/K;

        int ij = p*u;
        int ii = q*v;

        float sumval1 = 0;
        float sumval2 = 0;
        float sumval3 = 0;
        float sumval4 = 0;
        float sumval5 = 0;
        float sumval6 = 0;
        float sumval7 = 0;
        float sumval8 = 0;

         // input feature map
        
            for (unsigned int r = 0; r<R; r++) { // filter height
                for (unsigned int s = 0; s < S; s++) {// filter width
                    for(unsigned int c=0; c<C; c++) {
                
                    //output_seq[n][k][p][q] += input [n][c][ij+r][ii+s] * weight[k][c][r][s];
                    sumval1 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                    sumval2 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+c*R*S+r*S+s];
                    sumval3 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 2)*C*R*S+c*R*S+r*S+s];
                    sumval4 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 3)*C*R*S+c*R*S+r*S+s];
                    sumval5 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 4)*C*R*S+c*R*S+r*S+s];
                    sumval6 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 5)*C*R*S+c*R*S+r*S+s];
                    sumval7 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 6)*C*R*S+c*R*S+r*S+s];
                    sumval8 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 7)*C*R*S+c*R*S+r*S+s];
                }
            }
        }

        d_output[n*K*P*Q + k*P*Q + p*Q + q] = sumval1;
        d_output[n*K*P*Q + (k + 1)*P*Q + p*Q + q] = sumval2;
        d_output[n*K*P*Q + (k + 2)*P*Q + p*Q + q] = sumval3;
        d_output[n*K*P*Q + (k + 3)*P*Q + p*Q + q] = sumval4;
        d_output[n*K*P*Q + (k + 4)*P*Q + p*Q + q] = sumval5;
        d_output[n*K*P*Q + (k + 5)*P*Q + p*Q + q] = sumval6;
        d_output[n*K*P*Q + (k + 6)*P*Q + p*Q + q] = sumval7;
        d_output[n*K*P*Q + (k + 7)*P*Q + p*Q + q] = sumval8;

    }

}
