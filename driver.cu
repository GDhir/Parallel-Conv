
#include <hip/hip_runtime.h>
#include <iostream>
// #include <cuda.h>
// #include <stdio.h>
//you can change the grid_size
//you can change the block_size
#define BLOCK_SIZE 128
__global__ void cnn(int N,int C,int K,int H,int W,int R, int S, int u, int v, int P, int Q,
               float *d_input, float * d_weight, float * d_output){
    //@@ cnn kernel design

    int idx = threadIdx.x + blockIdx.x*blockDim.x;

    if( N > 4 ) {

        if( idx < N*K*P*Q/16 ) {

                int q = idx%Q;
                int p = (idx/Q)%P;
                int k = ( ( 4*(idx/Q/P) )%K );
                int n = 4*( ( 4*(idx/Q/P) )/K );

                int ij = p*u;
                int ii = q*v;

                float sumval1 = 0;
                float sumval2 = 0;
                float sumval3 = 0;
                float sumval4 = 0;
                float sumval5 = 0;
                float sumval6 = 0;
                float sumval7 = 0;
                float sumval8 = 0;
                float sumval9 = 0;
                float sumval10 = 0;
                float sumval11 = 0;
                float sumval12 = 0;
                float sumval13 = 0;
                float sumval14 = 0;
                float sumval15 = 0;
                float sumval16 = 0;

                    for (unsigned int r = 0; r<R; r++) { // filter height
                        for (unsigned int s = 0; s < S; s++) {// filter width
                            for(unsigned int c=0; c<C; c++) { 
                            //output_seq[n][k][p][q] += input [n][c][ij+r][ii+s] * weight[k][c][r][s];
                            sumval1 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                            sumval2 += d_input[(n)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+c*R*S+r*S+s];
                            sumval3 += d_input[(n)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 2)*C*R*S+c*R*S+r*S+s];
                            sumval4 += d_input[(n)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 3)*C*R*S+c*R*S+r*S+s];
                            sumval5 += d_input[(n + 1)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k)*C*R*S+c*R*S+r*S+s];
                            sumval6 += d_input[(n + 1)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+c*R*S+r*S+s];
                            sumval7 += d_input[(n + 1)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 2)*C*R*S+c*R*S+r*S+s];
                            sumval8 += d_input[(n + 1)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 3)*C*R*S+c*R*S+r*S+s];
                            sumval9 += d_input[(n + 2)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k)*C*R*S+c*R*S+r*S+s];
                            sumval10 += d_input[(n + 2)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+c*R*S+r*S+s];
                            sumval11 += d_input[(n + 2)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 2)*C*R*S+c*R*S+r*S+s];
                            sumval12 += d_input[(n + 2)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 3)*C*R*S+c*R*S+r*S+s];
                            sumval13 += d_input[(n + 3)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k)*C*R*S+c*R*S+r*S+s];
                            sumval14 += d_input[(n + 3)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+c*R*S+r*S+s];
                            sumval15 += d_input[(n + 3)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 2)*C*R*S+c*R*S+r*S+s];
                            sumval16 += d_input[(n + 3)*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 3)*C*R*S+c*R*S+r*S+s];

                        }
                    }
                }

                d_output[n*K*P*Q + k*P*Q + p*Q + q] = sumval1;
                d_output[(n)*K*P*Q + (k + 1)*P*Q + p*Q + q] = sumval2;
                d_output[(n)*K*P*Q + (k + 2)*P*Q + p*Q + q] = sumval3;
                d_output[(n)*K*P*Q + (k + 3)*P*Q + p*Q + q] = sumval4;
                d_output[(n + 1)*K*P*Q + (k)*P*Q + p*Q + q] = sumval5;
                d_output[(n + 1)*K*P*Q + (k + 1)*P*Q + p*Q + q] = sumval6;
                d_output[(n + 1)*K*P*Q + (k + 2)*P*Q + p*Q + q] = sumval7;
                d_output[(n + 1)*K*P*Q + (k + 3)*P*Q + p*Q + q] = sumval8;
                d_output[(n + 2)*K*P*Q + (k)*P*Q + p*Q + q] = sumval9;
                d_output[(n + 2)*K*P*Q + (k + 1)*P*Q + p*Q + q] = sumval10;
                d_output[(n + 2)*K*P*Q + (k + 2)*P*Q + p*Q + q] = sumval11;
                d_output[(n + 2)*K*P*Q + (k + 3)*P*Q + p*Q + q] = sumval12;
                d_output[(n + 3)*K*P*Q + (k)*P*Q + p*Q + q] = sumval13;
                d_output[(n + 3)*K*P*Q + (k + 1)*P*Q + p*Q + q] = sumval14;
                d_output[(n + 3)*K*P*Q + (k + 2)*P*Q + p*Q + q] = sumval15;
                d_output[(n + 3)*K*P*Q + (k + 3)*P*Q + p*Q + q] = sumval16;
            }
        }

    else if( C < 256 ) { 

        if( idx < N*K*P*Q/8 ) {

            int q = idx%Q;
            int p = (idx/Q)%P;
            int k = ( ( 8*(idx/Q/P) )%K );
            int n = ( 8*(idx/Q/P) )/K;

            int ij = p*u;
            int ii = q*v;

            float sumval1 = 0;
            float sumval2 = 0;
            float sumval3 = 0;
            float sumval4 = 0;
            float sumval5 = 0;
            float sumval6 = 0;
            float sumval7 = 0;
            float sumval8 = 0;

            // input feature map
            for (unsigned int r = 0; r<R; r++) { // filter height
                for (unsigned int s = 0; s < S; s++) {// filter width
                    for(unsigned int c=0; c<C; c++) {
                        //output_seq[n][k][p][q] += input [n][c][ij+r][ii+s] * weight[k][c][r][s];
                        sumval1 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                        sumval2 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+c*R*S+r*S+s];
                        sumval3 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 2)*C*R*S+c*R*S+r*S+s];
                        sumval4 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 3)*C*R*S+c*R*S+r*S+s];
                        sumval5 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 4)*C*R*S+c*R*S+r*S+s];
                        sumval6 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 5)*C*R*S+c*R*S+r*S+s];
                        sumval7 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 6)*C*R*S+c*R*S+r*S+s];
                        sumval8 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 7)*C*R*S+c*R*S+r*S+s];
                    }
                }
            }

            d_output[n*K*P*Q + k*P*Q + p*Q + q] = sumval1;
            d_output[n*K*P*Q + (k + 1)*P*Q + p*Q + q] = sumval2;
            d_output[n*K*P*Q + (k + 2)*P*Q + p*Q + q] = sumval3;
            d_output[n*K*P*Q + (k + 3)*P*Q + p*Q + q] = sumval4;
            d_output[n*K*P*Q + (k + 4)*P*Q + p*Q + q] = sumval5;
            d_output[n*K*P*Q + (k + 5)*P*Q + p*Q + q] = sumval6;
            d_output[n*K*P*Q + (k + 6)*P*Q + p*Q + q] = sumval7;
            d_output[n*K*P*Q + (k + 7)*P*Q + p*Q + q] = sumval8;

        }
    }
    else {

        if( idx < N*K*P*Q ) {
            int q = idx%Q;
            int p = (idx/Q)%P;
            int k = (idx/Q/P)%K;
            int n = idx/Q/P/K;

            int ij = p*u;
            int ii = q*v;

            float sumval = 0;
            int remc{ C%16 };

            // for(unsigned int c=0; c<C; c++) { // input feature map
            for (unsigned int r = 0; r<R; r++) { // filter height
                for (unsigned int s = 0; s < S; s++) {// filter width

                    for( unsigned int c = 0; c < remc; c++ ) {
                        sumval += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                    }

                    for(unsigned int c = remc; c < C; c += 16) {
                //output_seq[n][k][p][q] += input [n][c][ij+r][ii+s] * weight[k][c][r][s];
                        sumval += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                        sumval += d_input[n*C*H*W + (c + 1)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 1)*R*S + r*S + s];
                        sumval += d_input[n*C*H*W + (c + 2)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 2)*R*S+r*S+s];
                        sumval += d_input[n*C*H*W + (c + 3)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 3)*R*S + r*S + s];
                        sumval += d_input[n*C*H*W + (c + 4)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 4)*R*S+r*S+s];
                        sumval += d_input[n*C*H*W + (c + 5)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 5)*R*S + r*S + s];
                        sumval += d_input[n*C*H*W + (c + 6)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 6)*R*S+r*S+s];
                        sumval += d_input[n*C*H*W + (c + 7)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 7)*R*S + r*S + s];
                        sumval += d_input[n*C*H*W + (c + 8)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 8)*R*S + r*S + s];
                        sumval += d_input[n*C*H*W + (c + 9)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 9)*R*S+r*S+s];
                        sumval += d_input[n*C*H*W + (c + 10)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 10)*R*S + r*S + s];
                        sumval += d_input[n*C*H*W + (c + 11)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 11)*R*S+r*S+s];
                        sumval += d_input[n*C*H*W + (c + 12)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 12)*R*S + r*S + s];
                        sumval += d_input[n*C*H*W + (c + 13)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 13)*R*S+r*S+s];
                        sumval += d_input[n*C*H*W + (c + 14)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 14)*R*S + r*S + s];
                        sumval += d_input[n*C*H*W + (c + 15)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 15)*R*S + r*S + s];
                    }
                }
            }

            d_output[idx] = sumval;

        }

    }

}
