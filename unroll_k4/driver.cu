
#include <hip/hip_runtime.h>
// #include <iostream>
// #include <cuda.h>
// #include <stdio.h>
//you can change the grid_size
//you can change the block_size
#define BLOCK_SIZE 128
__global__ void cnn(int N,int C,int K,int H,int W,int R, int S, int u, int v, int P, int Q,
               float *d_input, float * d_weight, float * d_output){
    //@@ cnn kernel design

    int idx = threadIdx.x + blockIdx.x*BLOCK_SIZE;

    // long int Nout = N*K*P*Q;

    if( idx < N*K*P*Q/4 ) {
        int q = idx%Q;
        int p = (idx/Q)%P;
        int k = ( ( 4*(idx/Q/P) )%K );
        int n = ( 4*(idx/Q/P) )/K;

        int ij = p*u;
        int ii = q*v;

        float sumval1 = 0;
        float sumval2 = 0;
        float sumval3 = 0;
        float sumval4 = 0;

         // input feature map
        for(unsigned int c=0; c<C; c++) {
            for (unsigned int r = 0; r<R; r++) { // filter height
                for (unsigned int s = 0; s < S; s++) {// filter width
                
                    //output_seq[n][k][p][q] += input [n][c][ij+r][ii+s] * weight[k][c][r][s];
                    sumval1 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                    sumval2 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+c*R*S+r*S+s];
                    sumval3 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 2)*C*R*S+c*R*S+r*S+s];
                    sumval4 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 3)*C*R*S+c*R*S+r*S+s];

                }
            }
        }

        d_output[n*K*P*Q + k*P*Q + p*Q + q] = sumval1;
        d_output[n*K*P*Q + (k + 1)*P*Q + p*Q + q] = sumval2;
        d_output[n*K*P*Q + (k + 2)*P*Q + p*Q + q] = sumval3;
        d_output[n*K*P*Q + (k + 3)*P*Q + p*Q + q] = sumval4;

    }

}
