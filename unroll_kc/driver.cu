
#include <hip/hip_runtime.h>
// #include <iostream>
// #include <cuda.h>
// #include <stdio.h>
//you can change the grid_size
//you can change the block_size
#define BLOCK_SIZE 1024
__global__ void cnn(int N,int C,int K,int H,int W,int R, int S, int u, int v, int P, int Q,
               float *d_input, float * d_weight, float * d_output){
    //@@ cnn kernel design

    int idx = threadIdx.x + blockIdx.x*BLOCK_SIZE;

    // long int Nout = N*K*P*Q;

    if( idx < N*K*P*Q/2 ) {
        int q = idx%Q;
        int p = (idx/Q)%P;
        int k = ( ( 2*(idx/Q/P) )%K );
        int n = ( 2*(idx/Q/P) )/K;

        int ij = p*u;
        int ii = q*v;

        float sumval1 = 0;
        float sumval2 = 0;
        int remc{ C%8 };

        // for(unsigned int c=0; c<C; c++) { // input feature map
        for (unsigned int r = 0; r<R; r++) { // filter height
            for (unsigned int s = 0; s < S; s++) {// filter width

                for( unsigned int c = 0; c < remc; c++ ) {
                    sumval1 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                    sumval2 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+c*R*S+r*S+s];
                }

                for(unsigned int c = remc; c < C; c += 8) {
            //output_seq[n][k][p][q] += input [n][c][ij+r][ii+s] * weight[k][c][r][s];
                    sumval1 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                    sumval1 += d_input[n*C*H*W + (c + 1)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 1)*R*S + r*S + s];
                    sumval1 += d_input[n*C*H*W + (c + 2)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 2)*R*S+r*S+s];
                    sumval1 += d_input[n*C*H*W + (c + 3)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 3)*R*S + r*S + s];
                    sumval1 += d_input[n*C*H*W + (c + 4)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 4)*R*S+r*S+s];
                    sumval1 += d_input[n*C*H*W + (c + 5)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 5)*R*S + r*S + s];
                    sumval1 += d_input[n*C*H*W + (c + 6)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 6)*R*S+r*S+s];
                    sumval1 += d_input[n*C*H*W + (c + 7)*H*W + (ij+r)*W + ii+s] * d_weight[k*C*R*S+ (c + 7)*R*S + r*S + s];

                    sumval2 += d_input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+c*R*S+r*S+s];
                    sumval2 += d_input[n*C*H*W + (c + 1)*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+ (c + 1)*R*S + r*S + s];
                    sumval2 += d_input[n*C*H*W + (c + 2)*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+ (c + 2)*R*S+r*S+s];
                    sumval2 += d_input[n*C*H*W + (c + 3)*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+ (c + 3)*R*S + r*S + s];
                    sumval2 += d_input[n*C*H*W + (c + 4)*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+ (c + 4)*R*S+r*S+s];
                    sumval2 += d_input[n*C*H*W + (c + 5)*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+ (c + 5)*R*S + r*S + s];
                    sumval2 += d_input[n*C*H*W + (c + 6)*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+ (c + 6)*R*S+r*S+s];
                    sumval2 += d_input[n*C*H*W + (c + 7)*H*W + (ij+r)*W + ii+s] * d_weight[(k + 1)*C*R*S+ (c + 7)*R*S + r*S + s];
                }
            }
        }


        d_output[n*K*P*Q + k*P*Q + p*Q + q] = sumval1;
        d_output[n*K*P*Q + (k + 1)*P*Q + p*Q + q] = sumval2;

    }

}
